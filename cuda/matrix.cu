#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright 2018 Ahmed Tarek.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "util.h"

/**
 * Matrix x Vector
 * 
 * Each block calculates a single row in the matrix
 */

extern "C"
__global__ void matrix_mul_vector(float* m, float* v, float* r, long columns) {
    __shared__ float cache[THREADS_PER_BLOCK];
    
    int row = blockIdx.x;
    
    /* starting index of the row to calculate */
    long i_row = row * columns;
    
    int loop = calcIterations(blockDim.x, columns);
    
    cache[threadIdx.x] = 0;
    
    /* make sure we cover the whole vector */
    for(int i=0; i < loop; i++) {
        int j = i * blockDim.x + threadIdx.x;
        
        if(j < columns)
            cache[threadIdx.x] += m[i_row + j] * v[j];
    }

    __syncthreads();

    sumBlock(cache, min((long)blockDim.x, columns));
    
    if(threadIdx.x == 0) {
        r[row] = cache[0];
//        if(blockIdx.x == 0)
//            printf("%d %d\n", blockDim.x, blockDim.y);            
//            printf("%f %d\n", v[v_size-1], blockIdx.x);
    }
}

extern "C"
__global__ void memset_single_float(float* a, long i, float v) {
    a[i] = v;
}

extern "C"
__global__ void accumulate_vector(float* a, float* b, long size) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(row < size)
        a[row] += b[row];
}

extern "C"
__global__ void matrix_mul_vector_slow(float* m, float* v, float* r, long m_row, long v_size) {
    __shared__ float cache[THREADS_PER_BLOCK];
    
    for(int row=0; row < m_row; row++) {
        /* starting index of the row to calculate */
        long i_row = row * v_size;

        int loop = calcIterations(blockDim.x, v_size);

        cache[threadIdx.x] = 0;

        /* make sure we cover the whole vector */
        for(int i=0; i < loop; i++) {
            int j = i * blockDim.x + threadIdx.x;

            if(j < v_size)
                cache[threadIdx.x] += m[i_row + j] * v[j];
        }

        __syncthreads();

        sumBlock(cache, min((long)blockDim.x, v_size));

        if(threadIdx.x == 0) {
            r[row] = cache[0];
        }

        __syncthreads();
    }
}

/**
 * Matrix x Vector
 * 
 * Each block calculates a single row in the matrix
 */
extern "C"
__global__ void multi_matrix_mul_vector(float* m, long pitch_m, float* v, long pitch_v, float* r, long pitch_r, long columns) {
    __shared__ float cache[THREADS_PER_BLOCK];
    
    int row = blockIdx.x;
    
    /* starting index of the row to calculate */
    long i_row = row * columns;
    
    long m_base = blockIdx.y * pitch_m;
    long v_base = blockIdx.y * pitch_v;
    
    int loop = calcIterations(blockDim.x, columns);
    
    cache[threadIdx.x] = 0;
    
    /* make sure we cover the whole vector */
    for(int i=0; i < loop; i++) {
        int j = i * blockDim.x + threadIdx.x;
        
        if(j < columns)
            cache[threadIdx.x] += m[m_base + i_row + j] * v[v_base + j];
    }

    __syncthreads();

    sumBlock(cache, min((long)blockDim.x, columns));
        
    long r_base = blockIdx.y * pitch_r;
    
    if (threadIdx.x == 0)
        r[r_base + row] = cache[0];   
}

extern "C"
__global__ void multi_accumulate_vector(float* a, float* b, long size, size_t pitch) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int index = blockIdx.y * pitch + row;
    
    if(row < size)
        a[index] += b[index];
}
